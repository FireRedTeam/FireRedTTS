#include "hip/hip_runtime.h"
/* coding=utf-8
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include "type_shim.h"
#include <assert.h>
#include <cfloat>
#include <limits>
#include <stdint.h>
#include <c10/macros/Macros.h>

namespace {

    /*
template <typename Datatype, int ELEMENTS_PER_LDG>
__device__ __inline__ void copy_vector(Datatype *dst, const Datatype *src);

template <>
__device__ __inline__ void copy_vector<c10::BFloat16, 1>(c10::BFloat16 *dst, const c10::BFloat16 *src) { *dst = *src; }

template <>
__device__ __inline__ void copy_vector<c10::BFloat16, 4>(c10::BFloat16 *dst, const c10::BFloat16 *src) { *((float2*) dst) = *((float2*) src); }

template <>
__device__ __inline__ void copy_vector<c10::Half, 1>(c10::Half *dst, const c10::Half *src) { *dst = *src; }

template <>
__device__ __inline__ void copy_vector<c10::Half, 4>(c10::Half *dst, const c10::Half *src) { *((float2*) dst) = *((float2*) src); }

template <>
__device__ __inline__ void copy_vector<uint8_t, 1>(uint8_t *dst, const uint8_t *src) { *dst = *src; }

template <>
__device__ __inline__ void copy_vector<uint8_t, 4>(uint8_t *dst, const uint8_t *src) {*((half2*) dst) = *((half2*) src); }

int log2_ceil(int value) {
    int log2_value = 0;
    while ((1 << log2_value) < value) ++log2_value;
    return log2_value;
}

template<typename T>
struct Add {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a + b;
  }
};

template<typename T>
struct Max {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a < b ? b : a;
  }
};

template <typename T>
__device__ __forceinline__ T WARP_SHFL_XOR_NATIVE(T value, int laneMask, int width = warpSize, unsigned int mask = 0xffffffff)
{
#if CUDA_VERSION >= 9000
    return __shfl_xor_sync(mask, value, laneMask, width);
#else
    return __shfl_xor(value, laneMask, width);
#endif
}

template <typename acc_t, int WARP_BATCH, int WARP_SIZE, template<typename> class ReduceOp>
__device__ __forceinline__ void warp_reduce(acc_t* sum) {
    ReduceOp<acc_t> r;
    #pragma unroll
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        #pragma unroll
        for (int i = 0;  i < WARP_BATCH;  ++i) {
            acc_t b = WARP_SHFL_XOR_NATIVE(sum[i], offset, WARP_SIZE);
            sum[i] = r(sum[i], b);
        }
    }
}
*/

template <typename input_t, typename output_t, typename acc_t>
__global__ void anti_alias_activation_forward(
    output_t *dst,
    const input_t *src,
    const input_t *ftr,
    const input_t *alpha,
    const input_t *beta,
    int batch_size,
    int channels,
    int seq_len)
{
    // WARP_SIZE and WARP_BATCH must match the return values batches_per_warp and
    constexpr int ELEMENTS_PER_LDG_STG = 1; //(WARP_ITERATIONS < 4) ? 1 : 4;
    constexpr int BUFFER_SIZE = 32;
    constexpr int FILTER_SIZE = 12;
    constexpr int HALF_FILTER_SIZE = 6;
    constexpr int REPLICATION_PAD = 5; // 5 on each side

    // blockDim/threadIdx = (128, 1, 1)
    // gridDim/blockIdx = (seq_blocks, channels, batches)
    int block_offset = (blockIdx.x * 128 * BUFFER_SIZE + seq_len * (blockIdx.y + gridDim.y * blockIdx.z));
    int local_offset = threadIdx.x * BUFFER_SIZE;
    int seq_offset = blockIdx.x * 128 * BUFFER_SIZE + local_offset;


    //int intermediate_seq_len = seq_len * 2 - 1 + 4 * REPLICATION_PAD;
    //int intermediate_block_offset = (blockIdx.x * 128 * BUFFER_SIZE * 2 + intermediate_seq_len * (blockIdx.y + gridDim.y * blockIdx.z));
    //int intermediate_local_offset = threadIdx.x * BUFFER_SIZE * 2;

    int output_seq_len = seq_len * 2 ; //
    int output_block_offset = (blockIdx.x * 128 * BUFFER_SIZE * 2 + output_seq_len * (blockIdx.y + gridDim.y * blockIdx.z));
    int output_local_offset = threadIdx.x * BUFFER_SIZE * 2;
    int output_seq_offset = blockIdx.x * 128 * BUFFER_SIZE *2 + output_local_offset;
    // get values needed for replication padding before moving pointer
    const input_t *right_most_pntr = src + (seq_len * (blockIdx.y + gridDim.y * blockIdx.z));
    input_t seq_left_most_value = right_most_pntr[0];
    input_t seq_right_most_value = right_most_pntr[seq_len - 1];

    src += block_offset + local_offset;
    dst += output_block_offset + output_local_offset  ;
    alpha = alpha + blockIdx.y;
    input_t alpha_val = expf(alpha[0]);
    beta = beta + blockIdx.y;
    input_t beta_val = expf(beta[0]);
    // load data from global memory
    input_t elements[2*FILTER_SIZE+2*BUFFER_SIZE] = {0};
    input_t intermediates[2*FILTER_SIZE+2*BUFFER_SIZE] = {0};
    //output_t output[2*BUFFER_SIZE];
    input_t filter[FILTER_SIZE];
    //input_t temp_data[ELEMENTS_PER_LDG_STG];
    //uint8_t temp_mask[ELEMENTS_PER_LDG_STG];

    #pragma unroll
    for (int it = 0; it < FILTER_SIZE; it+=1) {
        filter[it] = ftr[it];
    }


    #pragma unroll
    for (int it = -HALF_FILTER_SIZE;  it < BUFFER_SIZE + HALF_FILTER_SIZE ;  it+=1) {
        int element_index = seq_offset + it;
	if ((element_index < 0) && (element_index >= -REPLICATION_PAD)) {
	    elements[2*(HALF_FILTER_SIZE+it)] = 2*seq_left_most_value;
	}
	if ((element_index >= seq_len) && (element_index < seq_len + REPLICATION_PAD)) {
	    elements[2*(HALF_FILTER_SIZE+it)] = 2*seq_right_most_value;
	}
        if ((element_index >= 0) && (element_index < seq_len)) {
	  elements[2*(HALF_FILTER_SIZE+it)] = 2*src[it];
        }
    }



    // apply filter
    #pragma unroll
    for (int it = 0;  it < (2 * BUFFER_SIZE + 2*FILTER_SIZE);  it+=1) {
        input_t acc = 0.0;

	int element_index = output_seq_offset + it; // index for output
	#pragma unroll
        for (int f_idx = 0; f_idx < FILTER_SIZE; f_idx+=1){
	  if ((element_index + f_idx) >= 0){
            acc += filter[f_idx] * elements[it+f_idx];
	  }
	}
        intermediates[it] = acc;
    }

    double no_div_by_zero = 0.000000001;
    #pragma unroll
    for (int it = 0; it < 12 + 2 * BUFFER_SIZE; it++) {
        intermediates[it] += (1.0/(beta_val + no_div_by_zero)) *  sinf(intermediates[it] * alpha_val) * sinf(intermediates[it] * alpha_val);
    }


    // now copy to output
    #pragma unroll
    for (int it = 0; it < 2*BUFFER_SIZE; it+=1){
      int element_index = output_seq_offset + it;
      if (element_index < output_seq_len) {
	dst[it]  = intermediates[it+6];
      }
    }



    // for (int it = 0;  it < BUFFER_SIZE;  it+=ELEMENTS_PER_LDG_STG) {
    //     int element_index = seq_offset + it;
    //     if (element_index < seq_len) {
    //         dst[it] = output[it];
    //     }
    // }


    // // Upsample convolution
    // for (int it = 0;  it < 2 * BUFFER_SIZE + 12;  it+=1) {
    //     input_t acc = 0.0;

    //     for (int f_idx = 0; f_idx < FILTER_SIZE; f_idx+=1){
    //         acc += filter[f_idx] * elements[it+f_idx];
    //     }
    //     intermediates[it] = acc;
    // }

    // // correct the corners of intermediates
    // if (seq_offset == 0) {
    //     for (int it = 0; it < 6; it+=1)
    //         intermediates[it] = 0;
    // }

    // if (seq_offset + 32 >= seq_len) {
    //     int offset = seq_len % 32 == 0 ? 32 : seq_len % 32;

    //     for (int it = 0; it < 6; it++) {
    //         intermediates[6+2*offset+it] = 0;
    //     }
    // }




    // for (int it = 0;  it < BUFFER_SIZE;  it+=ELEMENTS_PER_LDG_STG) {
    //     int element_index = seq_offset + it;
    //     if (element_index < seq_len) {
    //         dst[it] = output[it];
    //     }
    // }
}

template<typename input_t, typename output_t, typename acc_t>
void dispatch_anti_alias_activation_forward(
    output_t *dst,
    const input_t *src,
    const input_t *ftr,
    const input_t *alpha,
    const input_t *beta,
    int batch_size,
    int channels,
    int seq_len)
{
    if (seq_len == 0) {
        return;
    } else {
        // use 128 threads per block to maximimize gpu utilization
        constexpr int threads_per_block = 128;
        constexpr int seq_len_per_block = 4096;
        int blocks_per_seq_len = (seq_len + seq_len_per_block - 1) / seq_len_per_block;
        dim3 blocks(blocks_per_seq_len, channels, batch_size);
        dim3 threads(threads_per_block, 1, 1);

        anti_alias_activation_forward<input_t, output_t, acc_t>
	  <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(dst, src, ftr, alpha, beta, batch_size, channels, seq_len);
    }
}
}

namespace anti_alias_activation {

  torch::Tensor fwd_cuda(torch::Tensor const& input, torch::Tensor const& filter, torch::Tensor const& alpha, torch::Tensor const& beta)
{
  // input is a 4d tensor with dimensions [batches, attn_heads, seq_len, seq_len]
  const int batches = input.size(0);
  const int channels = input.size(1);
  const int seq_len = input.size(2);

  // Output
  auto act_options = input.options().requires_grad(false);
  int output_seq_len = seq_len*2; // we'll be dilating between each element by interspersing with zeros

  torch::Tensor anti_alias_activation_results =
      torch::empty({batches, channels, output_seq_len}, act_options);

  // Softmax Intermediate Result Ptr
  void* input_ptr = static_cast<void*>(input.data_ptr());
  void* filter_ptr = static_cast<void*>(filter.data_ptr());
  void* alpha_ptr = static_cast<void*>(alpha.data_ptr());
  void* beta_ptr = static_cast<void*>(beta.data_ptr());
  void* anti_alias_activation_results_ptr = static_cast<void*>(anti_alias_activation_results.data_ptr());

  DISPATCH_FLOAT_HALF_AND_BFLOAT(
      input.scalar_type(),
      "dispatch anti alias activation_forward",
      dispatch_anti_alias_activation_forward<scalar_t, scalar_t, float>(
        reinterpret_cast<scalar_t*>(anti_alias_activation_results_ptr),
	    reinterpret_cast<const scalar_t*>(input_ptr),
        reinterpret_cast<const scalar_t*>(filter_ptr),
        reinterpret_cast<const scalar_t*>(alpha_ptr),
	reinterpret_cast<const scalar_t*>(beta_ptr),
	    batches,
        channels,
        seq_len);
      );
  return anti_alias_activation_results;
}
}
